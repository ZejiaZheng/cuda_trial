#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include "timer.h"
#include "utils.h"

const int N = 1024;  // we are transposing a N by N mat
const int K = 16;

void fill_matrix(float* mat){
    for (int i = 0; i < N*N; i++){
        mat[i] = (float)i;
    }
}

int compare_matrices(float* mat1, float* mat2){
    for (int i = 0; i < N*N; i++){
        if(mat1[i] != mat2[i]) return 0;
    }
    return 1;
}

void transpose_cpu(float* mat, float* mat_new){
    for (int i = 0; i < N; i++){
        for (int j = 0; j< N; j++){
            mat_new[j*N + i] = mat[i*N + j];
        }
    }
}

__global__ void transpose_serial(float* mat, float* mat_new){
    for (int i = 0; i < N; i++){
        for (int j = 0; j< N; j++){
            mat_new[j*N + i] = mat[i*N + j];
        }
    }
}

__global__ void transpose_parallel_row(float* mat, float* mat_new){
    int i = threadIdx.x;

    for (int j = 0; j < N; j++){
        mat_new[j+i*N] = mat[i+j*N];
    }
}

__global__ void transpose_parallel_element(float* mat, float* mat_new){
    int i = blockIdx.x * K + threadIdx.x;
    int j = blockIdx.y * K + threadIdx.y;

    mat_new[j + i*N] = mat[i + j*N];
}

__global__ void transpose_parallel_element_tiling(float* mat, float* mat_new){
    int in_corner_i = blockIdx.x * K, in_corner_j = blockIdx.y * K;
    int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[K][K];

    tile[y][x] = mat[(in_corner_i+x) + (in_corner_j+y)*N];
    __syncthreads();

    mat_new[out_corner_i+x + (out_corner_j+y)*N] = tile[x][y];
}

int main(int argc, char const *argv[])
{
    size_t mat_size = N * N * sizeof(float);
    float* mat = (float *) malloc(mat_size);
    float* mat_new = (float *) malloc(mat_size);
    float* mat_gold = (float *) malloc(mat_size);

    fill_matrix(mat);

    clock_t start = clock();
    double diff;
    transpose_cpu(mat, mat_gold);
    diff = (double)(clock() - start);
    double msec = diff * 1000.0 / CLOCKS_PER_SEC;
    printf("CPU Transpose time taken: %f milliseconds \n", msec);


    float *d_in, *d_out;

    hipMalloc(&d_in, mat_size);
    hipMalloc(&d_out, mat_size);
    hipMemcpy(d_in, mat, mat_size, hipMemcpyHostToDevice);

    GpuTimer timer;
    
    // transpose serial
    timer.Start();
    transpose_serial<<<1,1>>>(d_in, d_out);
    timer.Stop();

    hipMemcpy(d_out, mat_new, mat_size, hipMemcpyDeviceToHost);

    printf("transpose_serial: %g ms.\nVerifying transpose...%s\n", timer.Elapsed(), compare_matrices(mat_new, mat_gold) ? "Failed" : "Success");

    // transpose parallel row
    timer.Start();
    transpose_parallel_row<<<1,N>>>(d_in, d_out);
    timer.Stop();

    hipMemcpy(d_out, mat_new, mat_size, hipMemcpyDeviceToHost);

    printf("transpose_parallel_row: %g ms.\nVerifying transpose...%s\n", timer.Elapsed(), compare_matrices(mat_new, mat_gold) ? "Failed" : "Success");

    // transpose parallel element
    dim3 blocks(N/K, N/K);
    dim3 threads(K,K);
    
    timer.Start();
    transpose_parallel_element<<<blocks,threads>>>(d_in, d_out);
    timer.Stop();

    hipMemcpy(d_out, mat_new, mat_size, hipMemcpyDeviceToHost);

    printf("transpose_parallel_element: %g ms.\nVerifying transpose...%s\n", timer.Elapsed(), compare_matrices(mat_new, mat_gold) ? "Failed" : "Success");

    // transpose by tiling
    timer.Start();
    transpose_parallel_element_tiling<<<blocks,threads>>>(d_in, d_out);
    timer.Stop();

    hipMemcpy(d_out, mat_new, mat_size, hipMemcpyDeviceToHost);

    printf("transpose_parallel_element_tiling: %g ms.\nVerifying transpose...%s\n", timer.Elapsed(), compare_matrices(mat_new, mat_gold) ? "Failed" : "Success");
    // show orignal mat, for debug purposes
    /*printf("original_mat: \n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%3.1f ", mat[i*N + j]);
        }
        printf("\n");
    }

    printf("transposed_mat: \n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%3.1f ", mat_gold[i*N + j]);
        }
        printf("\n");
    }*/
    hipFree(d_in);
    hipFree(d_out);
    free(mat);
    free(mat_gold);

    return 0;
}