#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include "timer.h"
#include "utils.h"

const int N = 1024;  // we are transposing a N by N mat
const int K = 1;

void fill_matrix(float* mat){
    for (int i = 0; i < N*N; i++){
        mat[i] = (float)i;
    }
}

void compare_matrices(float* mat1, float* mat2){
    for (int i = 0; i < N*N; i++){
        if(mat1[i] != mat2[i]) return 0;
    }
    return 1;
}

void transpose_cpu(float* mat, float* mat_new){
    for (int i = 0; i < N; i++){
        for (int j = 0; j< N; j++){
            mat_new[j*N + i] = mat[i*N + j];
        }
    }
}

void transpose_serial(float* mat, float* mat_new){
    for (int i = 0; i < N; i++){
        for (int j = 0; j< N; j++){
            mat_new[j*N + i] = mat[i*N + j];
        }
    }
}

int main(int argc, char const *argv[])
{
    size_t mat_size = N * N * sizeof(float);
    float* mat = (float *) malloc(mat_size);
    float* mat_new = (float *) malloc(mat_size);
    float* mat_gold = (float *) malloc(mat_size);

    fill_matrix(mat);

    clock_t start = clock(), diff;
    transpose_cpu(mat, mat_gold);
    diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("CPU Transpose time taken %d seconds, %d milliseconds \n", msec/1000, msec%1000);


    float *d_in, *d_out;

    hipMalloc(&d_in, mat_size);
    hipMalloc(&d_out, mat_size);
    hipMemcpy(d_in, mat, mat_size, hipMemcpyHostToDevice);

    GpuTimer timer;
    timer.Start();
    transpose_serial<<<1,1>>>(d_in, d_out);
    timer.Stop();

    hipMemcpy(d_out, mat_new, mat_size, hipMemcpyDeviceToHost);

    printf("transpose_parallel_per_row: %g ms.\nVerifying transpose...%s\n", timer.Elapsed(), compare_matrices(mat_new, mat_gold) ? "Failed" : "Success");

    

    // show orignal mat, for debug purposes
    /*printf("original_mat: \n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%3.1f ", mat[i*N + j]);
        }
        printf("\n");
    }

    printf("transposed_mat: \n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%3.1f ", mat_gold[i*N + j]);
        }
        printf("\n");
    }*/

    free(mat);
    free(mat_gold);

    return 0;
}